/*
* This program uses the device CURAND API to calculate what
* proportion of pseudo - random ints have low bit set.
*/
# include <stdio.h>
# include <stdlib.h>
# include <hip/hip_runtime.h>
# include "hiprand/hiprand_kernel.h"
# include <vector>

# define CUDA_CALL(x) do { if ((x) != hipSuccess ) {	\
printf (" Error at %s:%d\n", __FILE__ , __LINE__ );		\
return EXIT_FAILURE ;}} while (0)						\

__global__ void setup_kernel ( hiprandState * state )
{
//	int id = threadIdx .x + blockIdx .x * c_thread;
	int x = threadIdx.x + blockIdx.x*blockDim.x ; 
	int y = threadIdx.y + blockIdx.y*blockDim.y ;
	int offset = x+y*blockDim.x*gridDim.x;
	hiprand_init (1234 , offset, 0, & state [offset]);
}

__global__ void generate_kernel ( hiprandState* state , float* result )
{
	int x = threadIdx.x + blockIdx.x*blockDim.x ; 
	int y = threadIdx.y + blockIdx.y*blockDim.y ;
	int offset = x+y*blockDim.x*gridDim.x;

	hiprandState localState = state [offset ];
//	result [offset] = curand_normal (& localState );
	result [offset] = offset;
}

__global__ void look( int* threadIdxx
					, int* threadIdxy
					, int* blockIdxx
					, int* blockIdxy
					, int* blockDimx
					, int* blockDimy
					, int* gridDimx
					, int* gridDimy
					, int* doffset
					)
{
	int x = threadIdx.x + blockIdx.x*blockDim.x ; 
	int y = threadIdx.y + blockIdx.y*blockDim.y ;
	int offset = x+y*blockDim.x*gridDim.x;

	threadIdxx[offset]=threadIdx.x;
	threadIdxy[offset]=threadIdx.y;
	blockIdxx[offset] = blockIdx.x;
	blockIdxy[offset] = blockIdx.y;
	blockDimx [offset] = blockDim.x ;
	blockDimy [offset] = blockDim.y ;
	gridDimx [offset]= gridDim.x;
	gridDimy [offset]= gridDim.y;
	doffset[offset]= offset;
}



int main (int argc , char * argv [])
{
	int i;
	hiprandState * devStates ;
	float * devResults , * hostResults ;
	
	int gridsize = 5;
	int blocksize  = 2;

	int num = gridsize*blocksize*blocksize;

	dim3 block(gridsize);
	dim3 threads(blocksize,blocksize);

	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop,0);
	// ideally we can use 8,5 Million threads to generate the random numbers

	hostResults = (float *) calloc (num, sizeof ( float));

	CUDA_CALL ( hipMalloc (( void **)& devResults , num* sizeof ( float)));
	CUDA_CALL ( hipMemset ( devResults , 0, num * sizeof (float)) );
	CUDA_CALL ( hipMalloc (( void **)& devStates , num *sizeof ( hiprandState )));

	int  *dthreadIdxx
		, *dthreadIdxy
		, *dblockIdxx
		, *dblockIdxy
		, *dblockDimx
		, *dblockDimy
		, *dgridDimx
		, *dgridDimy
		, *doffset;

	CUDA_CALL ( hipMalloc (( void **)& dthreadIdxx , num* sizeof ( int)));
	CUDA_CALL ( hipMalloc (( void **)& dthreadIdxy , num* sizeof ( int)));
	CUDA_CALL ( hipMalloc (( void **)& dblockIdxx , num* sizeof ( int)));
	CUDA_CALL ( hipMalloc (( void **)& dblockIdxy , num* sizeof ( int)));
	CUDA_CALL ( hipMalloc (( void **)& dblockDimx , num* sizeof ( int)));
	CUDA_CALL ( hipMalloc (( void **)& dblockDimy , num* sizeof ( int)));
	CUDA_CALL ( hipMalloc (( void **)& dgridDimx , num* sizeof ( int)));
	CUDA_CALL ( hipMalloc (( void **)& dgridDimy , num* sizeof ( int)));
	CUDA_CALL ( hipMalloc (( void **)& doffset , num* sizeof ( int)));

	int   *hthreadIdxx = (int *) calloc (num, sizeof ( int));
	int   *hthreadIdxy = (int *) calloc (num, sizeof ( int));
	int   *hblockIdxx = (int *) calloc (num, sizeof ( int));
	int   *hblockIdxy = (int *) calloc (num, sizeof ( int));
	int   *hblockDimx = (int *) calloc (num, sizeof ( int));
	int   *hblockDimy = (int *) calloc (num, sizeof ( int));
	int   *hgridDimx = (int *) calloc (num, sizeof ( int));
	int   *hgridDimy = (int *) calloc (num, sizeof ( int));
	int   *hoffset = (int *) calloc (num, sizeof ( int));


	look<<<block,threads>>>(	  dthreadIdxx
								, dthreadIdxy
								, dblockIdxx
								, dblockIdxy
								, dblockDimx
								, dblockDimy
								, dgridDimx
								, dgridDimy
								, doffset);


	CUDA_CALL ( hipMemcpy ( hthreadIdxx , dthreadIdxx, num *sizeof (float), hipMemcpyDeviceToHost ));
	CUDA_CALL ( hipMemcpy ( hthreadIdxy , dthreadIdxy, num *sizeof (float), hipMemcpyDeviceToHost ));
	CUDA_CALL ( hipMemcpy ( hblockIdxx  , dblockIdxx , num *sizeof (float), hipMemcpyDeviceToHost ));
	CUDA_CALL ( hipMemcpy ( hblockIdxy  , dblockIdxy , num *sizeof (float), hipMemcpyDeviceToHost ));
	CUDA_CALL ( hipMemcpy ( hblockDimx  , dblockDimx , num *sizeof (float), hipMemcpyDeviceToHost ));
	CUDA_CALL ( hipMemcpy ( hblockDimy  , dblockDimy , num *sizeof (float), hipMemcpyDeviceToHost ));
	CUDA_CALL ( hipMemcpy ( hgridDimx	 , dgridDimx  , num *sizeof (float), hipMemcpyDeviceToHost ));
	CUDA_CALL ( hipMemcpy ( hgridDimy   , dgridDimy  , num *sizeof (float), hipMemcpyDeviceToHost ));
	CUDA_CALL ( hipMemcpy ( hoffset	 , doffset	  , num *sizeof (float), hipMemcpyDeviceToHost ));

	std::vector<int> sthreadIdxx(hthreadIdxx,hthreadIdxx+num)
				   , sthreadIdxy(hthreadIdxy,hthreadIdxy+num)
				   , sblockIdxx(hblockIdxx,hblockIdxx+num)
				   , sblockIdxy(hblockIdxy,hblockIdxy+num)
				   , sblockDimx(hblockDimx,hblockDimx+num)
				   , sblockDimy(hblockDimy,hblockDimy+num)
				   , sgridDimx(hgridDimx,hgridDimx+num)
				   , sgridDimy(hgridDimy,hgridDimy+num)
				   , soffset(hoffset,hoffset+num);

	/* Cleanup */
	CUDA_CALL ( hipFree ( dthreadIdxx));
	CUDA_CALL ( hipFree ( dthreadIdxy));
	CUDA_CALL ( hipFree ( dblockIdxx));
	CUDA_CALL ( hipFree ( dblockIdxy));
	CUDA_CALL ( hipFree ( dblockDimx));
	CUDA_CALL ( hipFree ( dblockDimy));
	CUDA_CALL ( hipFree ( dgridDimx));
	CUDA_CALL ( hipFree ( dgridDimy));


	free ( hostResults );

	free ( hthreadIdxx);
	free ( hthreadIdxy);
	free ( hblockIdxx);
	free ( hblockIdxy);
	free ( hblockDimx);
	free ( hblockDimy);
	free ( hgridDimx);
	free ( hgridDimy);

	return EXIT_SUCCESS ;
}